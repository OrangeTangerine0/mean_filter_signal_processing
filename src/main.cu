#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <cmath>
#include "signal_utils.h"

#define BLOCK_SIZE 256
#define WINDOW_SIZE 5

__global__ void meanFilter(const float* input, float* output, int length) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        float sum = 0;
        int count = 0;
        for (int j = -WINDOW_SIZE / 2; j <= WINDOW_SIZE / 2; ++j) {
            int pos = idx + j;
            if (pos >= 0 && pos < length) {
                sum += input[pos];
                count++;
            }
        }
        output[idx] = sum / count;
    }
}

void processFile(const char* filename, const char* outname) {
    float* h_data;
    int length = 0;
    loadCSV(filename, nullptr, &length);
    h_data = new float[length];
    loadCSV(filename, h_data, &length);

    float* d_input;
    float* d_output;
    hipMalloc(&d_input, length * sizeof(float));
    hipMalloc(&d_output, length * sizeof(float));

    hipMemcpy(d_input, h_data, length * sizeof(float), hipMemcpyHostToDevice);

    int blocks = (length + BLOCK_SIZE - 1) / BLOCK_SIZE;
    meanFilter<<<blocks, BLOCK_SIZE>>>(d_input, d_output, length);

    hipMemcpy(h_data, d_output, length * sizeof(float), hipMemcpyDeviceToHost);

    interpolate(h_data, length);
    saveCSV(outname, h_data, length);

    hipFree(d_input);
    hipFree(d_output);
    delete[] h_data;
}

int main() {
    processFile("data/sample_signal.csv", "output/mean_filtered_signal.csv");
    std::cout << "Processing completed.\n";
    return 0;
}
